/* Performance testing for assignment operator '+='
Comparison between '+=' and '='
Two kernels should be executed separately or else the kernels will influence each other
Kernel that executed later was faster regardless of operator
Result :	 assignment operator '=' is faster than '+='
			100 ms vs 94 ms for 256*100000  nodes
This result will be beneficial to optimize Peridynamics kernel

*/

#include <hip/hip_runtime.h>
#include <cmath>	//for calculating power & NaN
#include<iostream>
#include<cstdio>
#include <vector>
#include <cstdlib>
#include <fstream> // for writing to file
#include <math.h>       //exp, pi
#include <chrono>	//for time measurement
#include <fstream>
#include <ctime>

using namespace std;
using namespace std::chrono;

//Use assignment operator '+='
__global__ void add_01 (float *array_a,
	 float *array_b,
	 float *array_c ){
	
	size_t i= threadIdx.x+ blockDim.x*threadIdx.y+blockDim.x*blockDim.y*blockIdx.x;
	
	array_c[i] += (array_a[i]+array_b[i])*sqrt(array_a[i])*sqrt(array_b[i]);
}

//Use assignment operator '='
__global__ void add_02 (float *array_a,
	 float *array_b,
	 float *array_c ){
	
	size_t i= threadIdx.x+ blockDim.x*threadIdx.y+blockDim.x*blockDim.y*blockIdx.x;
	
	array_c[i] = (array_a[i]+array_b[i])*sqrt(array_a[i])*sqrt(array_b[i]);
	
}


int main(int argc, char **argv){
	cout<<"Start of program assignment operator test"<<endl;
	const size_t node = 256*100000;
	
	//Differentiate a,b to 01 & 02 to avoid kernel reusing the value
	float *array_a_01= (float*) calloc (node, sizeof(float));
	float *array_b_01= (float*) calloc (node, sizeof(float));
	float *array_a_02= (float*) calloc (node, sizeof(float));
	float *array_b_02= (float*) calloc (node, sizeof(float));
	float *array_c_01= (float*) calloc (node, sizeof(float));
	float *array_c_02= (float*) calloc (node, sizeof(float));

	cout<<"No of nodes = "<<node<<endl;
	
	//Initialization
	for (size_t i = 0; i < node; ++i) {
			array_a_01[i]= 2.30*i; // dummy value
			array_b_01[i]= 1.45*i; // dummy value
			
			array_a_02 [i] = array_a_01 [i];
			array_b_02 [i] = array_b_01 [i];
	}
	
	//###########################################################################
	/*
	int blockSize;      // The launch configurator returned block size 
    int minGridSize;    // The minimum grid size needed to achieve the maximum occupancy for a full device launch 
    int gridSize;       // The actual grid size needed, based on input size 
	
	cudaOccupancyMaxPotentialBlockSize(&minGridSize, &blockSize, cal_dilatation, 0, node); 
	gridSize = (node + blockSize - 1) / blockSize; 
	
	printf("\t Blocksize= %i\n", blockSize);
	printf("\t minGridSize= %i \n",minGridSize);
	printf("\t gridSize= %i \n",gridSize);*/
	
	
	// Create memory buffers on the device for each vector 
	
	float* buffer_array_a_01, * buffer_array_b_01;
	float* buffer_array_a_02, * buffer_array_b_02;
	float * buffer_array_c_01, * buffer_array_c_02;
	
	hipMalloc(&buffer_array_a_01, node*sizeof(float));
	hipMalloc(&buffer_array_a_02, node*sizeof(float));
	hipMalloc(&buffer_array_b_01, node*sizeof(float));
	hipMalloc(&buffer_array_b_02, node*sizeof(float));
	hipMalloc(&buffer_array_c_01, node*sizeof(float));
	hipMalloc(&buffer_array_c_02, node*sizeof(float));
	
	hipMemcpy(buffer_array_a_01, array_a_01, node*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_a_02, array_a_02, node*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_b_01, array_b_01, node*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_b_02, array_b_02, node*sizeof(float), hipMemcpyHostToDevice);
	
	dim3 gridDim(node/256,1,1);         // 512 x 1 x 1
	dim3 blockDim(256, 1, 1); // 1024 x 1024 x 1
	
	
	printf("Using manual gridDim %i, ", gridDim.x);
	printf("blockDim %i, ", blockDim.x);
	printf("= %i;", (gridDim.x*blockDim.x));
	printf("\t Gap = %zi \n", (gridDim.x*blockDim.x)-node);
	
	
	/*printf("Using optimized blockSize %i, ", blockSize);
	printf("gridSize %i, ", gridSize);
	printf("= %i;", (gridSize*blockSize));
	printf("\t Gap = %zi \n", (gridSize*blockSize)-node);*/
	
	//Comment out one of the 2 kernels below---------------------------	
	
/*
	system_clock::time_point start_01 = system_clock::now();
	add_01<<< gridDim , blockDim>>>(buffer_array_a_01, buffer_array_b_01, buffer_array_c_01);
	cudaMemcpy(array_c_01, buffer_array_c_01, node *sizeof(float), cudaMemcpyDeviceToHost);
	cudaDeviceSynchronize();
	system_clock::time_point stop_01 = system_clock::now();
	
	std::chrono::duration<float, std::milli> duration_01 = stop_01 - start_01;
	cout << "Assignment operator '+=' time  = "<<duration_01.count()<<" millisecond"<<endl;
	*/
	//------------------------------------------------------------
	

	system_clock::time_point start_02 = system_clock::now();
	add_02<<< gridDim , blockDim>>>(buffer_array_a_02, buffer_array_b_02, buffer_array_c_02);
	hipMemcpy(array_c_02, buffer_array_c_02, node *sizeof(float), hipMemcpyDeviceToHost);
	hipDeviceSynchronize();
	system_clock::time_point stop_02 = system_clock::now();
	
	std::chrono::duration<float, std::milli> duration_02 = stop_02 - start_02;
	cout << "Assignment operator '=' time  = "<<duration_02.count()<<" millisecond"<<endl;
	
	
	
   //##########################################################
	
	//Checking
	
	
	printf("\t array_c_01 - array_c_02 : \n");
	for (size_t i = 0; i < 5; ++i) {
		printf("\t\t %f", array_c_01[i]);	
		printf(" - %f \n",  array_c_02[i]);	
	}
	
	
	hipFree(buffer_array_a_01); hipFree(buffer_array_a_02);
	hipFree(buffer_array_b_01); hipFree(buffer_array_b_02); 
	hipFree(buffer_array_c_01); 
	hipFree(buffer_array_c_02); 
	
	free(array_a_01);	free(array_a_02);
	free (array_b_01);	free(array_b_02);	
	free (array_c_01);  free (array_c_02);
	
	printf("End of program!\n\n");
	
	
}