/*
Test max arg size (256 Byte) for Cuda kernel.
Passing struct to kernel arguments to overcome limitation of number of args in kernel
But regular kernel turns out working properly with 70 args

https://devtalk.nvidia.com/default/topic/458705/is-there-any-limit-on-of-arguments-in-cuda-kernel-/

*/

#include <hip/hip_runtime.h>
#include <cmath>	//for calculating power & NaN
#include<iostream>
#include<cstdio>
#include <vector>
#include <cstdlib>
#include <fstream> // for writing to file
#include <math.h>       //exp, pi
#include <chrono>	//for time measurement
#include <fstream>
#include <ctime>


using namespace std;
using namespace std::chrono;


struct struct_ab {
	double* array_01;	double* array_02;	double* array_03;	
	double* array_04;	double* array_05;	double* array_06;
	double* array_07;	double* array_08;	double* array_09;		
	double* array_10;	double* array_11;	double* array_12;
	double* array_13;	double* array_14;	double* array_15;
	double* array_16;	double* array_17;	double* array_18;
	double* array_19;	double* array_20;	double* array_21;
	double* array_22;	double* array_23;	double* array_24;
	double* array_25;	double* array_26;	double* array_27;
	double* array_28;	double* array_29;	double* array_30;
	double* array_31;	double* array_32;	double* array_33;
	double* array_34;	double* array_35;	double* array_36;
	double* array_37;	double* array_38;	double* array_39;
	double* array_40;	double* array_41;	double* array_42;
	double* array_43;	double* array_44;	double* array_45;
	double* array_46;	double* array_47;	double* array_48;
	double* array_49;	double* array_50;	
	double* array_51;	double* array_52;	double* array_53;	
	double* array_54;	double* array_55;	double* array_56;
	double* array_57;	double* array_58;	double* array_59;		
	double* array_60;
	double* array_61;	double* array_62;	double* array_63;	
	double* array_64;	double* array_65;	double* array_66;
	double* array_67;	double* array_68;	double* array_69;		
	double* array_70;	
	};


__global__ void vector_add_struct(struct_ab mystruct_ab, double* c){

	size_t i= threadIdx.x+ blockDim.x*threadIdx.y+blockDim.x*blockDim.y*blockIdx.x;
	c[i]=mystruct_ab.array_01[i]+mystruct_ab.array_02[i]\
		+ mystruct_ab.array_03[i]+mystruct_ab.array_04[i]\
		+ mystruct_ab.array_05[i]+mystruct_ab.array_06[i]\
		+ mystruct_ab.array_07[i]+mystruct_ab.array_08[i]\
		+ mystruct_ab.array_09[i]+mystruct_ab.array_10[i]\
		+ mystruct_ab.array_11[i]+mystruct_ab.array_12[i]\
		+ mystruct_ab.array_13[i]+mystruct_ab.array_14[i]\
		+ mystruct_ab.array_15[i]+mystruct_ab.array_16[i]\
		+ mystruct_ab.array_17[i]+mystruct_ab.array_18[i]\
		+ mystruct_ab.array_19[i]+mystruct_ab.array_20[i]\
		+ mystruct_ab.array_21[i]+mystruct_ab.array_22[i]\
		+ mystruct_ab.array_23[i]+mystruct_ab.array_24[i]\
		+ mystruct_ab.array_25[i]+mystruct_ab.array_26[i]\
		+ mystruct_ab.array_27[i]+mystruct_ab.array_28[i]\
		+ mystruct_ab.array_29[i]+mystruct_ab.array_30[i]\
		+ mystruct_ab.array_31[i]+mystruct_ab.array_32[i]\
		+ mystruct_ab.array_33[i]+mystruct_ab.array_34[i]\
		+ mystruct_ab.array_35[i]+mystruct_ab.array_36[i]\
		+ mystruct_ab.array_37[i]+mystruct_ab.array_38[i]\
		+ mystruct_ab.array_39[i]+mystruct_ab.array_40[i]\
		+ mystruct_ab.array_41[i]+mystruct_ab.array_42[i]\
		+ mystruct_ab.array_43[i]+mystruct_ab.array_44[i]\
		+ mystruct_ab.array_45[i]+mystruct_ab.array_46[i]\
		+ mystruct_ab.array_47[i]+mystruct_ab.array_48[i]\
		+ mystruct_ab.array_49[i]+mystruct_ab.array_50[i]\
		+ mystruct_ab.array_51[i]+mystruct_ab.array_52[i]\
		+ mystruct_ab.array_53[i]+mystruct_ab.array_54[i]\
		+ mystruct_ab.array_55[i]+mystruct_ab.array_56[i]\
		+ mystruct_ab.array_57[i]+mystruct_ab.array_58[i]\
		+ mystruct_ab.array_59[i]+mystruct_ab.array_60[i]\
		+ mystruct_ab.array_61[i]+mystruct_ab.array_62[i]\
		+ mystruct_ab.array_63[i]+mystruct_ab.array_64[i]\
		+ mystruct_ab.array_65[i]+mystruct_ab.array_66[i]\
		+ mystruct_ab.array_67[i]+mystruct_ab.array_68[i]\
		+ mystruct_ab.array_69[i]+mystruct_ab.array_70[i]
	;
}

//max = 43 args 
__global__ void vector_add70(double* array_01, double* array_02, double* array_03, 
		double* array_04, double* array_05, double* array_06, 
		double* array_07, double* array_08, double* array_09,
		double* array_10,
		double* array_11, double* array_12, double* array_13, 
		double* array_14, double* array_15, double* array_16, 
		double* array_17, double* array_18, double* array_19,
		double* array_20,
		double* array_21, double* array_22, double* array_23, 
		double* array_24, double* array_25, double* array_26, 
		double* array_27, double* array_28, double* array_29,
		double* array_30,
		double* array_31, double* array_32, double* array_33, 
		double* array_34, double* array_35, double* array_36,	
		double* array_37, double* array_38, double* array_39,
		double* array_40,
		double* array_41, double* array_42, double* array_43, 
		double* array_44, double* array_45, double* array_46, 
		double* array_47, double* array_48, double* array_49,
		double* array_50,
		double* array_51, double* array_52, double* array_53, 
		double* array_54, double* array_55, double* array_56, 
		double* array_57, double* array_58, double* array_59,
		double* array_60,
		double* array_61, double* array_62, double* array_63, 
		double* array_64, double* array_65, double* array_66, 
		double* array_67, double* array_68, double* array_69,
		double* array_70,
		
		double* c){

	size_t i= threadIdx.x+ blockDim.x*threadIdx.y+blockDim.x*blockDim.y*blockIdx.x;
	c[i]=c[i]=array_01[i]+array_02[i] + array_03[i]+array_04[i] 
		+ array_05[i]+array_06[i] + array_07[i]+array_08[i]\
		+ array_09[i]+array_10[i]\
		+ array_11[i]+array_12[i] + array_13[i]+array_14[i]\
		+ array_15[i]+array_16[i] + array_17[i]+array_18[i]\
		+ array_19[i]+array_20[i]\
		+ array_21[i]+array_22[i] + array_23[i]+array_24[i]\
		+ array_25[i]+array_26[i] + array_27[i]+array_28[i]\
		+ array_29[i]+array_30[i]\
		+ array_31[i]+array_32[i] + array_33[i]+array_34[i]\
		+ array_35[i]+array_36[i] + array_37[i]+array_38[i]\
		+ array_39[i]+array_40[i]\
		+ array_41[i]+array_42[i] + array_43[i]+array_44[i]\
		+ array_45[i]+array_46[i] + array_47[i]+array_48[i]\
		+ array_49[i]+array_50[i]
		+ array_51[i]+array_52[i] + array_53[i]+array_54[i] 
		+ array_55[i]+array_56[i] + array_57[i]+array_58[i]\
		+ array_59[i]+array_60[i]\
		+ array_61[i]+array_62[i] + array_63[i]+array_64[i] 
		+ array_65[i]+array_66[i] + array_67[i]+array_68[i]\
		+ array_69[i]+array_70[i]\
		
	;
}

int main(int argc, char **argv){
	cout<<"Start of program"<<endl;
	const size_t node = 100;
		
	double *array_01_serial = new double[node]; double *array_02_serial = new double[node];
	double *array_03_serial = new double[node]; double *array_04_serial = new double[node];
	double *array_05_serial = new double[node]; double *array_06_serial = new double[node];
	double *array_07_serial = new double[node]; double *array_08_serial = new double[node];
	double *array_09_serial = new double[node]; double *array_10_serial = new double[node];
	double *array_11_serial = new double[node]; double *array_12_serial = new double[node];
	double *array_13_serial = new double[node]; double *array_14_serial = new double[node];
	double *array_15_serial = new double[node]; double *array_16_serial = new double[node];
	double *array_17_serial = new double[node]; double *array_18_serial = new double[node];
	double *array_19_serial = new double[node]; double *array_20_serial = new double[node];
	double *array_21_serial = new double[node]; double *array_22_serial = new double[node];
	double *array_23_serial = new double[node]; double *array_24_serial = new double[node];
	double *array_25_serial = new double[node]; double *array_26_serial = new double[node];
	double *array_27_serial = new double[node]; double *array_28_serial = new double[node];
	double *array_29_serial = new double[node]; double *array_30_serial = new double[node];
	double *array_31_serial = new double[node]; double *array_32_serial = new double[node];
	double *array_33_serial = new double[node]; double *array_34_serial = new double[node];
	double *array_35_serial = new double[node]; double *array_36_serial = new double[node];
	double *array_37_serial = new double[node]; double *array_38_serial = new double[node];
	double *array_39_serial = new double[node]; double *array_40_serial = new double[node];
	double *array_41_serial = new double[node]; double *array_42_serial = new double[node];
	double *array_43_serial = new double[node]; double *array_44_serial = new double[node];
	double *array_45_serial = new double[node]; double *array_46_serial = new double[node];
	double *array_47_serial = new double[node]; double *array_48_serial = new double[node];
	double *array_49_serial = new double[node]; double *array_50_serial = new double[node];
	double *array_51_serial = new double[node]; double *array_52_serial = new double[node];
	double *array_53_serial = new double[node]; double *array_54_serial = new double[node];
	double *array_55_serial = new double[node]; double *array_56_serial = new double[node];
	double *array_57_serial = new double[node]; double *array_58_serial = new double[node];
	double *array_59_serial = new double[node]; double *array_60_serial = new double[node];
	double *array_61_serial = new double[node]; double *array_62_serial = new double[node];
	double *array_63_serial = new double[node]; double *array_64_serial = new double[node];
	double *array_65_serial = new double[node]; double *array_66_serial = new double[node];
	double *array_67_serial = new double[node]; double *array_68_serial = new double[node];
	double *array_69_serial = new double[node]; double *array_70_serial = new double[node];
	
	
	double *c_serial = new double[node];
	
	double *array_01 = new double[node];	double *array_02 = new double[node];
	double *array_03 = new double[node]; 	double *array_04 = new double[node];
	double *array_05 = new double[node];	double *array_06 = new double[node];
	double *array_07 = new double[node];	double *array_08 = new double[node];	
	double *array_09 = new double[node];	double *array_10 = new double[node];	
	double *array_11 = new double[node];	double *array_12 = new double[node];	
	double *array_13 = new double[node];	double *array_14 = new double[node];	
	double *array_15 = new double[node];	double *array_16 = new double[node];	
	double *array_17 = new double[node];	double *array_18 = new double[node];	
	double *array_19 = new double[node];	double *array_20 = new double[node];
	double *array_21 = new double[node];	double *array_22 = new double[node];
	double *array_23 = new double[node];	double *array_24 = new double[node];
	double *array_25 = new double[node];	double *array_26 = new double[node];
	double *array_27 = new double[node];	double *array_28 = new double[node];
	double *array_29 = new double[node];	double *array_30 = new double[node];
	double *array_31 = new double[node];	double *array_32 = new double[node];
	double *array_33 = new double[node];	double *array_34 = new double[node];
	double *array_35 = new double[node];	double *array_36 = new double[node];
	double *array_37 = new double[node];	double *array_38 = new double[node];	
	double *array_39 = new double[node];	double *array_40 = new double[node];	
	double *array_41 = new double[node];	double *array_42 = new double[node];
	double *array_43 = new double[node]; 	double *array_44 = new double[node];
	double *array_45 = new double[node];	double *array_46 = new double[node];
	double *array_47 = new double[node];	double *array_48 = new double[node];	
	double *array_49 = new double[node];	double *array_50 = new double[node];	
	double *array_51 = new double[node];	double *array_52 = new double[node];
	double *array_53 = new double[node]; 	double *array_54 = new double[node];
	double *array_55 = new double[node];	double *array_56 = new double[node];
	double *array_57 = new double[node];	double *array_58 = new double[node];	
	double *array_59 = new double[node];	double *array_60 = new double[node];	
	double *array_61 = new double[node];	double *array_62 = new double[node];
	double *array_63 = new double[node]; 	double *array_64 = new double[node];
	double *array_65 = new double[node];	double *array_66 = new double[node];
	double *array_67 = new double[node];	double *array_68 = new double[node];	
	double *array_69 = new double[node];	double *array_70 = new double[node];	
	
	
	
	double *c = new double[node];
	
	cout<<"No of nodes = "<<node<<endl;
	
	//Initialization
	for (size_t i = 0; i < node; ++i) {
			std::srand(std::time(nullptr)); // use current time as seed for random generator
			array_01[i]=((double) rand()/(RAND_MAX))*i;	array_02[i]=((double) rand()/(RAND_MAX))*i;
			array_03[i]=((double) rand()/(RAND_MAX))*i;	array_04[i]=((double) rand()/(RAND_MAX))*i;
			array_05[i]=((double) rand()/(RAND_MAX))*i;	array_06[i]=((double) rand()/(RAND_MAX))*i;
			array_07[i]=((double) rand()/(RAND_MAX))*i;	array_08[i]=((double) rand()/(RAND_MAX))*i;
			array_09[i]=((double) rand()/(RAND_MAX))*i;	array_10[i]=((double) rand()/(RAND_MAX))*i;
			
			array_11[i]=((double) rand()/(RAND_MAX))*i;	array_12[i]=((double) rand()/(RAND_MAX))*i;
			array_13[i]=((double) rand()/(RAND_MAX))*i;	array_14[i]=((double) rand()/(RAND_MAX))*i;
			array_15[i]=((double) rand()/(RAND_MAX))*i;	array_16[i]=((double) rand()/(RAND_MAX))*i;
			array_17[i]=((double) rand()/(RAND_MAX))*i;	array_18[i]=((double) rand()/(RAND_MAX))*i;
			array_19[i]=((double) rand()/(RAND_MAX))*i;	array_20[i]=((double) rand()/(RAND_MAX))*i;
			
			array_21[i]=((double) rand()/(RAND_MAX))*i;	array_22[i]=((double) rand()/(RAND_MAX))*i;
			array_23[i]=((double) rand()/(RAND_MAX))*i;	array_24[i]=((double) rand()/(RAND_MAX))*i;
			array_25[i]=((double) rand()/(RAND_MAX))*i;	array_26[i]=((double) rand()/(RAND_MAX))*i;
			array_27[i]=((double) rand()/(RAND_MAX))*i;	array_28[i]=((double) rand()/(RAND_MAX))*i;
			array_29[i]=((double) rand()/(RAND_MAX))*i;	array_30[i]=((double) rand()/(RAND_MAX))*i;
		
			array_31[i]=((double) rand()/(RAND_MAX))*i;	array_32[i]=((double) rand()/(RAND_MAX))*i;
			array_33[i]=((double) rand()/(RAND_MAX))*i;	array_34[i]=((double) rand()/(RAND_MAX))*i;
			array_35[i]=((double) rand()/(RAND_MAX))*i;	array_36[i]=((double) rand()/(RAND_MAX))*i;
			array_37[i]=((double) rand()/(RAND_MAX))*i;	array_38[i]=((double) rand()/(RAND_MAX))*i;
			array_39[i]=((double) rand()/(RAND_MAX))*i;	array_40[i]=((double) rand()/(RAND_MAX))*i;
			
			array_41[i]=((double) rand()/(RAND_MAX))*i;	array_42[i]=((double) rand()/(RAND_MAX))*i;
			array_43[i]=((double) rand()/(RAND_MAX))*i;	array_44[i]=((double) rand()/(RAND_MAX))*i;
			array_45[i]=((double) rand()/(RAND_MAX))*i;	array_46[i]=((double) rand()/(RAND_MAX))*i;
			array_47[i]=((double) rand()/(RAND_MAX))*i;	array_48[i]=((double) rand()/(RAND_MAX))*i;
			array_49[i]=((double) rand()/(RAND_MAX))*i;	array_50[i]=((double) rand()/(RAND_MAX))*i;
			array_51[i]=((double) rand()/(RAND_MAX))*i;	array_52[i]=((double) rand()/(RAND_MAX))*i;
			array_53[i]=((double) rand()/(RAND_MAX))*i;	array_54[i]=((double) rand()/(RAND_MAX))*i;
			array_55[i]=((double) rand()/(RAND_MAX))*i;	array_56[i]=((double) rand()/(RAND_MAX))*i;
			array_57[i]=((double) rand()/(RAND_MAX))*i;	array_58[i]=((double) rand()/(RAND_MAX))*i;
			array_59[i]=((double) rand()/(RAND_MAX))*i;	array_60[i]=((double) rand()/(RAND_MAX))*i;
			array_61[i]=((double) rand()/(RAND_MAX))*i;	array_62[i]=((double) rand()/(RAND_MAX))*i;
			array_63[i]=((double) rand()/(RAND_MAX))*i;	array_64[i]=((double) rand()/(RAND_MAX))*i;
			array_65[i]=((double) rand()/(RAND_MAX))*i;	array_66[i]=((double) rand()/(RAND_MAX))*i;
			array_67[i]=((double) rand()/(RAND_MAX))*i;	array_68[i]=((double) rand()/(RAND_MAX))*i;
			array_69[i]=((double) rand()/(RAND_MAX))*i;	array_70[i]=((double) rand()/(RAND_MAX))*i;
			
		
			//-------------------------------------------
			array_01_serial[i]= array_01[i];	array_02_serial[i]= array_02[i];			
			array_03_serial[i]= array_03[i];	array_04_serial[i]= array_04[i];
			array_05_serial[i]= array_05[i];	array_06_serial[i]= array_06[i];
			array_07_serial[i]= array_07[i];	array_08_serial[i]= array_08[i];
			array_09_serial[i]= array_09[i];	array_10_serial[i]= array_10[i];
			array_11_serial[i]= array_11[i];	array_12_serial[i]= array_12[i];
			array_13_serial[i]= array_13[i];	array_14_serial[i]= array_14[i];
			array_15_serial[i]= array_15[i];	array_16_serial[i]= array_16[i];
			array_17_serial[i]= array_17[i];	array_18_serial[i]= array_18[i];
			array_19_serial[i]= array_19[i];	array_20_serial[i]= array_20[i];
			array_21_serial[i]= array_21[i];	array_22_serial[i]= array_22[i];
			array_23_serial[i]= array_23[i];	array_24_serial[i]= array_24[i];
			array_25_serial[i]= array_25[i];	array_26_serial[i]= array_26[i];
			array_27_serial[i]= array_27[i];	array_28_serial[i]= array_28[i];
			array_29_serial[i]= array_29[i];	array_30_serial[i]= array_30[i];
			array_31_serial[i]= array_31[i];	array_32_serial[i]= array_32[i];
			array_33_serial[i]= array_33[i];	array_34_serial[i]= array_34[i];
			array_35_serial[i]= array_35[i];	array_36_serial[i]= array_36[i];			
			array_37_serial[i]= array_37[i];	array_38_serial[i]= array_38[i];
			array_39_serial[i]= array_39[i];	array_40_serial[i]= array_40[i];
			
			array_41_serial[i]= array_41[i];	array_42_serial[i]= array_42[i];			
			array_43_serial[i]= array_43[i];	array_44_serial[i]= array_44[i];
			array_45_serial[i]= array_45[i];	array_46_serial[i]= array_46[i];
			array_47_serial[i]= array_47[i];	array_48_serial[i]= array_48[i];
			array_49_serial[i]= array_49[i];	array_50_serial[i]= array_50[i];
			array_51_serial[i]= array_51[i];	array_52_serial[i]= array_52[i];			
			array_53_serial[i]= array_53[i];	array_54_serial[i]= array_54[i];
			array_55_serial[i]= array_55[i];	array_56_serial[i]= array_56[i];
			array_57_serial[i]= array_57[i];	array_58_serial[i]= array_58[i];
			array_59_serial[i]= array_59[i];	array_60_serial[i]= array_60[i];
			array_61_serial[i]= array_61[i];	array_62_serial[i]= array_62[i];			
			array_63_serial[i]= array_63[i];	array_64_serial[i]= array_64[i];
			array_65_serial[i]= array_65[i];	array_66_serial[i]= array_66[i];
			array_67_serial[i]= array_67[i];	array_68_serial[i]= array_68[i];
			array_69_serial[i]= array_69[i];	array_70_serial[i]= array_70[i];
			
		}

//	#########################################################
//Serial version
	for (size_t i = 0; i < node; ++i) {
		c_serial[i]=array_01_serial[i]+array_02_serial[i] + array_03_serial[i]+array_04_serial[i]\
			+ array_05_serial[i]+array_06_serial[i]	+ array_07_serial[i]+array_08_serial[i]\
			+ array_09_serial[i]+array_10_serial[i]\
			+ array_11_serial[i]+array_12_serial[i] + array_13_serial[i]+array_14_serial[i]\
			+ array_15_serial[i]+array_16_serial[i]	+ array_17_serial[i]+array_18_serial[i]\
			+ array_19_serial[i]+array_20_serial[i]\
			+ array_21_serial[i]+array_22_serial[i]	+ array_23_serial[i]+array_24_serial[i]\
			+ array_25_serial[i]+array_26_serial[i]	+ array_27_serial[i]+array_28_serial[i]\
			+ array_29_serial[i]+array_30_serial[i]\
			+ array_31_serial[i]+array_32_serial[i]	+ array_33_serial[i]+array_34_serial[i]\
			+ array_35_serial[i]+array_36_serial[i]	+ array_37_serial[i]+array_38_serial[i]\
			+ array_39_serial[i]+array_40_serial[i]\
			+ array_41_serial[i]+array_42_serial[i]	+ array_43_serial[i]+array_44_serial[i]\
			+ array_45_serial[i]+array_46_serial[i]	+ array_47_serial[i]+array_48_serial[i]\
			+ array_49_serial[i]+array_50_serial[i]\
			+ array_51_serial[i]+array_52_serial[i] + array_53_serial[i]+array_54_serial[i]\
			+ array_55_serial[i]+array_56_serial[i]	+ array_57_serial[i]+array_58_serial[i]\
			+ array_59_serial[i]+array_60_serial[i]\
			+ array_61_serial[i]+array_62_serial[i] + array_63_serial[i]+array_64_serial[i]\
			+ array_65_serial[i]+array_66_serial[i]	+ array_67_serial[i]+array_68_serial[i]\
			+ array_69_serial[i]+array_70_serial[i]\
			
		;
	}
	
	//###########################################################################
	//PARALLEL VERSION
	cout<<"Start parallel version"<<endl;
	
	// Create memory buffers on the device for each vector 
	double* buffer_array_01, *buffer_array_02, *buffer_array_03;
	double* buffer_array_04, *buffer_array_05, *buffer_array_06; 
	double* buffer_array_07, *buffer_array_08, *buffer_array_09;
	double* buffer_array_10;
	double* buffer_array_11, *buffer_array_12, *buffer_array_13;
	double* buffer_array_14, *buffer_array_15, *buffer_array_16; 
	double* buffer_array_17, *buffer_array_18, *buffer_array_19;
	double* buffer_array_20;
	double* buffer_array_21, *buffer_array_22, *buffer_array_23;
	double* buffer_array_24, *buffer_array_25, *buffer_array_26; 
	double* buffer_array_27, *buffer_array_28, *buffer_array_29;
	double* buffer_array_30;
	double* buffer_array_31, *buffer_array_32, *buffer_array_33;
	double* buffer_array_34, *buffer_array_35, *buffer_array_36; 
	double* buffer_array_37, *buffer_array_38, *buffer_array_39;
	double* buffer_array_40;
	double* buffer_array_41, *buffer_array_42, *buffer_array_43;
	double* buffer_array_44, *buffer_array_45, *buffer_array_46; 
	double* buffer_array_47, *buffer_array_48, *buffer_array_49;
	double* buffer_array_50;
	double* buffer_array_51, *buffer_array_52, *buffer_array_53;
	double* buffer_array_54, *buffer_array_55, *buffer_array_56; 
	double* buffer_array_57, *buffer_array_58, *buffer_array_59;
	double* buffer_array_60;
	double* buffer_array_61, *buffer_array_62, *buffer_array_63;
	double* buffer_array_64, *buffer_array_65, *buffer_array_66; 
	double* buffer_array_67, *buffer_array_68, *buffer_array_69;
	double* buffer_array_70;
	
	double *buffer_c;
	
	struct_ab mystruct_ab_d; //struct in device
	
	hipMalloc(&buffer_array_01, node*sizeof(double));
	hipMalloc(&buffer_array_02, node*sizeof(double));
	hipMalloc(&buffer_array_03, node*sizeof(double));
	hipMalloc(&buffer_array_04, node*sizeof(double));
	hipMalloc(&buffer_array_05, node*sizeof(double));
	hipMalloc(&buffer_array_06, node*sizeof(double));
	hipMalloc(&buffer_array_07, node*sizeof(double));	
	hipMalloc(&buffer_array_08, node*sizeof(double));
	hipMalloc(&buffer_array_09, node*sizeof(double));
	hipMalloc(&buffer_array_10, node*sizeof(double));
	hipMalloc(&buffer_array_11, node*sizeof(double));
	hipMalloc(&buffer_array_12, node*sizeof(double));
	hipMalloc(&buffer_array_13, node*sizeof(double));
	hipMalloc(&buffer_array_14, node*sizeof(double));
	hipMalloc(&buffer_array_15, node*sizeof(double));
	hipMalloc(&buffer_array_16, node*sizeof(double));
	hipMalloc(&buffer_array_17, node*sizeof(double));	
	hipMalloc(&buffer_array_18, node*sizeof(double));
	hipMalloc(&buffer_array_19, node*sizeof(double));
	hipMalloc(&buffer_array_20, node*sizeof(double));
	hipMalloc(&buffer_array_21, node*sizeof(double));
	hipMalloc(&buffer_array_22, node*sizeof(double));
	hipMalloc(&buffer_array_23, node*sizeof(double));
	hipMalloc(&buffer_array_24, node*sizeof(double));
	hipMalloc(&buffer_array_25, node*sizeof(double));
	hipMalloc(&buffer_array_26, node*sizeof(double));
	hipMalloc(&buffer_array_27, node*sizeof(double));	
	hipMalloc(&buffer_array_28, node*sizeof(double));
	hipMalloc(&buffer_array_29, node*sizeof(double));
	hipMalloc(&buffer_array_30, node*sizeof(double));
	hipMalloc(&buffer_array_31, node*sizeof(double));
	hipMalloc(&buffer_array_32, node*sizeof(double));
	hipMalloc(&buffer_array_33, node*sizeof(double));
	hipMalloc(&buffer_array_34, node*sizeof(double));
	hipMalloc(&buffer_array_35, node*sizeof(double));
	hipMalloc(&buffer_array_36, node*sizeof(double));
	hipMalloc(&buffer_array_37, node*sizeof(double));	
	hipMalloc(&buffer_array_38, node*sizeof(double));
	hipMalloc(&buffer_array_39, node*sizeof(double));
	hipMalloc(&buffer_array_40, node*sizeof(double));
	hipMalloc(&buffer_array_41, node*sizeof(double));
	hipMalloc(&buffer_array_42, node*sizeof(double));
	hipMalloc(&buffer_array_43, node*sizeof(double));
	hipMalloc(&buffer_array_44, node*sizeof(double));
	hipMalloc(&buffer_array_45, node*sizeof(double));
	hipMalloc(&buffer_array_46, node*sizeof(double));
	hipMalloc(&buffer_array_47, node*sizeof(double));	
	hipMalloc(&buffer_array_48, node*sizeof(double));
	hipMalloc(&buffer_array_49, node*sizeof(double));
	hipMalloc(&buffer_array_50, node*sizeof(double));
	hipMalloc(&buffer_array_51, node*sizeof(double));
	hipMalloc(&buffer_array_52, node*sizeof(double));
	hipMalloc(&buffer_array_53, node*sizeof(double));
	hipMalloc(&buffer_array_54, node*sizeof(double));
	hipMalloc(&buffer_array_55, node*sizeof(double));
	hipMalloc(&buffer_array_56, node*sizeof(double));
	hipMalloc(&buffer_array_57, node*sizeof(double));	
	hipMalloc(&buffer_array_58, node*sizeof(double));
	hipMalloc(&buffer_array_59, node*sizeof(double));
	hipMalloc(&buffer_array_60, node*sizeof(double));
	hipMalloc(&buffer_array_61, node*sizeof(double));
	hipMalloc(&buffer_array_62, node*sizeof(double));
	hipMalloc(&buffer_array_63, node*sizeof(double));
	hipMalloc(&buffer_array_64, node*sizeof(double));
	hipMalloc(&buffer_array_65, node*sizeof(double));
	hipMalloc(&buffer_array_66, node*sizeof(double));
	hipMalloc(&buffer_array_67, node*sizeof(double));	
	hipMalloc(&buffer_array_68, node*sizeof(double));
	hipMalloc(&buffer_array_69, node*sizeof(double));
	hipMalloc(&buffer_array_70, node*sizeof(double));
	
	
	hipMalloc(&buffer_c, node*sizeof(double));		
	
	hipMemcpy(buffer_array_01, array_01, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_02, array_02, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_03, array_03, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_04, array_04, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_05, array_05, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_06, array_06, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_07, array_07, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_08, array_08, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_09, array_09, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_10, array_10, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_11, array_11, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_12, array_12, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_13, array_13, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_14, array_14, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_15, array_15, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_16, array_16, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_17, array_17, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_18, array_18, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_19, array_19, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_20, array_20, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_21, array_21, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_22, array_22, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_23, array_23, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_24, array_24, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_25, array_25, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_26, array_26, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_27, array_27, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_28, array_28, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_29, array_29, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_30, array_30, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_31, array_31, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_32, array_32, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_33, array_33, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_34, array_34, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_35, array_35, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_36, array_36, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_37, array_37, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_38, array_38, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_39, array_39, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_40, array_40, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_41, array_41, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_42, array_42, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_43, array_43, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_44, array_44, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_45, array_45, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_46, array_46, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_47, array_47, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_48, array_48, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_49, array_49, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_50, array_50, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_51, array_51, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_52, array_52, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_53, array_53, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_54, array_54, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_55, array_55, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_56, array_56, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_57, array_57, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_58, array_58, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_59, array_59, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_60, array_60, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_61, array_61, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_62, array_62, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_63, array_63, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_64, array_64, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_65, array_65, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_66, array_66, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_67, array_67, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_68, array_68, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_69, array_69, node*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(buffer_array_70, array_70, node*sizeof(double), hipMemcpyHostToDevice);
	

	mystruct_ab_d.array_01 = buffer_array_01; mystruct_ab_d.array_02 = buffer_array_02; 
	mystruct_ab_d.array_03 = buffer_array_03; mystruct_ab_d.array_04 = buffer_array_04; 
	mystruct_ab_d.array_05 = buffer_array_05; mystruct_ab_d.array_06 = buffer_array_06; 
	mystruct_ab_d.array_07 = buffer_array_07; mystruct_ab_d.array_08 = buffer_array_08; 
	mystruct_ab_d.array_09 = buffer_array_09; mystruct_ab_d.array_10 = buffer_array_10; 
	mystruct_ab_d.array_11 = buffer_array_11; mystruct_ab_d.array_12 = buffer_array_12; 
	mystruct_ab_d.array_13 = buffer_array_13; mystruct_ab_d.array_14 = buffer_array_14; 
	mystruct_ab_d.array_15 = buffer_array_15; mystruct_ab_d.array_16 = buffer_array_16; 
	mystruct_ab_d.array_17 = buffer_array_17; mystruct_ab_d.array_18 = buffer_array_18; 
	mystruct_ab_d.array_19 = buffer_array_19; mystruct_ab_d.array_20 = buffer_array_20; 
	mystruct_ab_d.array_21 = buffer_array_21; mystruct_ab_d.array_22 = buffer_array_22; 
	mystruct_ab_d.array_23 = buffer_array_23; mystruct_ab_d.array_24 = buffer_array_24; 
	mystruct_ab_d.array_25 = buffer_array_25; mystruct_ab_d.array_26 = buffer_array_26; 
	mystruct_ab_d.array_27 = buffer_array_27; mystruct_ab_d.array_28 = buffer_array_28; 
	mystruct_ab_d.array_29 = buffer_array_29; mystruct_ab_d.array_30 = buffer_array_30; 
	mystruct_ab_d.array_31 = buffer_array_31; mystruct_ab_d.array_32 = buffer_array_32; 
	mystruct_ab_d.array_33 = buffer_array_33; mystruct_ab_d.array_34 = buffer_array_34; 
	mystruct_ab_d.array_35 = buffer_array_35; mystruct_ab_d.array_36 = buffer_array_36; 
	mystruct_ab_d.array_37 = buffer_array_37; mystruct_ab_d.array_38 = buffer_array_38; 
	mystruct_ab_d.array_39 = buffer_array_39; mystruct_ab_d.array_40 = buffer_array_40; 
	mystruct_ab_d.array_41 = buffer_array_41; mystruct_ab_d.array_42 = buffer_array_42; 
	mystruct_ab_d.array_43 = buffer_array_43; mystruct_ab_d.array_44 = buffer_array_44; 
	mystruct_ab_d.array_45 = buffer_array_45; mystruct_ab_d.array_46 = buffer_array_46; 
	mystruct_ab_d.array_47 = buffer_array_47; mystruct_ab_d.array_48 = buffer_array_48; 
	mystruct_ab_d.array_49 = buffer_array_49; mystruct_ab_d.array_50 = buffer_array_50; 
	mystruct_ab_d.array_51 = buffer_array_51; mystruct_ab_d.array_52 = buffer_array_52; 
	mystruct_ab_d.array_53 = buffer_array_53; mystruct_ab_d.array_54 = buffer_array_54; 
	mystruct_ab_d.array_55 = buffer_array_55; mystruct_ab_d.array_56 = buffer_array_56; 
	mystruct_ab_d.array_57 = buffer_array_57; mystruct_ab_d.array_58 = buffer_array_58; 
	mystruct_ab_d.array_59 = buffer_array_59; mystruct_ab_d.array_60 = buffer_array_60; 
	mystruct_ab_d.array_61 = buffer_array_61; mystruct_ab_d.array_62 = buffer_array_62; 
	mystruct_ab_d.array_63 = buffer_array_63; mystruct_ab_d.array_64 = buffer_array_64; 
	mystruct_ab_d.array_65 = buffer_array_65; mystruct_ab_d.array_66 = buffer_array_66; 
	mystruct_ab_d.array_67 = buffer_array_67; mystruct_ab_d.array_68 = buffer_array_68; 
	mystruct_ab_d.array_69 = buffer_array_69; mystruct_ab_d.array_70 = buffer_array_70; 
	
	
	
	
	//Use manual block size
	dim3 gridDim(node,1,1);         // 512 x 512 x 64
	dim3 blockDim(1, 1, 1); // 1024 x 1024 x 64, max threads/block=128-512 (multiple of 32)
	
	
	cout<<"Not using struct"<<endl;		
	vector_add70<<<gridDim,blockDim>>>(buffer_array_01, buffer_array_02, buffer_array_03, 
		buffer_array_04, buffer_array_05, buffer_array_06, 
		buffer_array_07, buffer_array_08, buffer_array_09, 
		buffer_array_10, 
		buffer_array_11, buffer_array_12, buffer_array_13, 
		buffer_array_14, buffer_array_15, buffer_array_16, 
		buffer_array_17, buffer_array_18, buffer_array_19, 
		buffer_array_20, 
		buffer_array_21, buffer_array_22, buffer_array_23, 
		buffer_array_24, buffer_array_25, buffer_array_26, 
		buffer_array_27, buffer_array_28, buffer_array_29, 
		buffer_array_30, 
		buffer_array_31, buffer_array_32, buffer_array_33, 
		buffer_array_34, buffer_array_35, buffer_array_36, 		
		buffer_array_37, buffer_array_38, buffer_array_39, 
		buffer_array_40, 
		buffer_array_41, buffer_array_42, buffer_array_43, 
		buffer_array_44, buffer_array_45, buffer_array_46, 
		buffer_array_47, buffer_array_48, buffer_array_49, 
		buffer_array_50,	
		buffer_array_51, buffer_array_52, buffer_array_53, 
		buffer_array_54, buffer_array_55, buffer_array_56, 
		buffer_array_57, buffer_array_58, buffer_array_59, 
		buffer_array_60, 
		buffer_array_61, buffer_array_62, buffer_array_63, 
		buffer_array_64, buffer_array_65, buffer_array_66, 
		buffer_array_67, buffer_array_68, buffer_array_69, 
		buffer_array_70, 		
		
		buffer_c); 
	
	/*
	cout<<"Using struct"<<endl;
	vector_add_struct<<<gridDim,blockDim>>>(mystruct_ab_d, buffer_c); 
	*/
	
	hipDeviceSynchronize();
	hipMemcpy(c, buffer_c, node *sizeof(double), hipMemcpyDeviceToHost);
	
   //##########################################################
	
	//Checking
	double sum_c_serial=0.0;
	double sum_c=0.0;
	
	for (size_t i = 0; i < 6; ++i) {
		cout<< "\t"<< c[i] <<" - "<< c_serial[i] << endl;
	}
	
	for (size_t i = 0; i < node; ++i) {
		sum_c += c[i];
		sum_c_serial += c_serial[i];
	}
	cout<<"Gap= "<<sum_c - sum_c_serial <<endl;
	
	hipFree(buffer_array_01);	hipFree(buffer_array_02);
	hipFree(buffer_array_03);	hipFree(buffer_array_04);
	hipFree(buffer_array_05);	hipFree(buffer_array_06);
	hipFree(buffer_array_07);	hipFree(buffer_array_08);
	hipFree(buffer_array_09);	hipFree(buffer_array_10);
	hipFree(buffer_array_11);	hipFree(buffer_array_12);
	hipFree(buffer_array_13);	hipFree(buffer_array_14);
	hipFree(buffer_array_15);	hipFree(buffer_array_16);
	hipFree(buffer_array_17);	hipFree(buffer_array_18);
	hipFree(buffer_array_19);	hipFree(buffer_array_20);
	hipFree(buffer_array_21);	hipFree(buffer_array_22);
	hipFree(buffer_array_23);	hipFree(buffer_array_24);
	hipFree(buffer_array_25);	hipFree(buffer_array_26);
	hipFree(buffer_array_27);	hipFree(buffer_array_28);
	hipFree(buffer_array_29);	hipFree(buffer_array_30);
	hipFree(buffer_array_31);	hipFree(buffer_array_32);
	hipFree(buffer_array_33);	hipFree(buffer_array_34);
	hipFree(buffer_array_35);	hipFree(buffer_array_36);
	hipFree(buffer_array_37);	hipFree(buffer_array_38);
	hipFree(buffer_array_39);	hipFree(buffer_array_40);
	hipFree(buffer_array_41);	hipFree(buffer_array_42);
	hipFree(buffer_array_43);	hipFree(buffer_array_44);
	hipFree(buffer_array_45);	hipFree(buffer_array_46);
	hipFree(buffer_array_47);	hipFree(buffer_array_48);
	hipFree(buffer_array_49);	hipFree(buffer_array_50);
	hipFree(buffer_array_51);	hipFree(buffer_array_52);
	hipFree(buffer_array_53);	hipFree(buffer_array_54);
	hipFree(buffer_array_55);	hipFree(buffer_array_56);
	hipFree(buffer_array_57);	hipFree(buffer_array_58);
	hipFree(buffer_array_59);	hipFree(buffer_array_60);
	hipFree(buffer_array_61);	hipFree(buffer_array_62);
	hipFree(buffer_array_63);	hipFree(buffer_array_64);
	hipFree(buffer_array_65);	hipFree(buffer_array_66);
	hipFree(buffer_array_67);	hipFree(buffer_array_68);
	hipFree(buffer_array_69);	hipFree(buffer_array_70);
	
	hipFree(buffer_c);	
	
	delete [] array_01;	delete [] array_02;	delete [] array_03;	delete [] array_04;	
	delete [] array_05;	delete [] array_06;	delete [] array_07;	delete [] array_08;	
	delete [] array_09;	delete [] array_10;	
	delete [] array_11;	delete [] array_12;	delete [] array_13;	delete [] array_14;	
	delete [] array_15;	delete [] array_16;	delete [] array_17;	delete [] array_18;	
	delete [] array_19;	delete [] array_20;	
	delete [] array_21;	delete [] array_22;	delete [] array_23;	delete [] array_24;	
	delete [] array_25;	delete [] array_26;	delete [] array_27;	delete [] array_28;	
	delete [] array_29;	delete [] array_30;	
	delete [] array_31;	delete [] array_32;	delete [] array_33;	delete [] array_34;	
	delete [] array_35;	delete [] array_36;	delete [] array_37;	delete [] array_38;	
	delete [] array_39;	delete [] array_40;
	delete [] array_41;	delete [] array_42;	delete [] array_43;	delete [] array_44;	
	delete [] array_45;	delete [] array_46;	delete [] array_47;	delete [] array_48;	
	delete [] array_49;	delete [] array_50;	
	delete [] array_51;	delete [] array_52;	delete [] array_53;	delete [] array_54;	
	delete [] array_55;	delete [] array_56;	delete [] array_57;	delete [] array_58;	
	delete [] array_59;	delete [] array_60;
	delete [] array_61;	delete [] array_62;	delete [] array_63;	delete [] array_64;	
	delete [] array_65;	delete [] array_66;	delete [] array_67;	delete [] array_68;	
	delete [] array_69;	delete [] array_70;
	
	delete [] c;
	
	delete [] array_01_serial;	delete [] array_02_serial;	delete [] array_03_serial;	
	delete [] array_04_serial; 	delete [] array_05_serial;	delete [] array_06_serial;
	delete [] array_07_serial;	delete [] array_08_serial; delete [] array_09_serial;
	delete [] array_10_serial;
	delete [] array_11_serial;	delete [] array_12_serial;	delete [] array_13_serial;	
	delete [] array_14_serial; 	delete [] array_15_serial;	delete [] array_16_serial;
	delete [] array_17_serial;	delete [] array_18_serial; delete [] array_19_serial;
	delete [] array_20_serial;
	delete [] array_21_serial;	delete [] array_22_serial;	delete [] array_23_serial;	
	delete [] array_24_serial; 	delete [] array_25_serial;	delete [] array_26_serial;
	delete [] array_27_serial;	delete [] array_28_serial; delete [] array_29_serial;
	delete [] array_30_serial;
	delete [] array_31_serial;	delete [] array_32_serial;	delete [] array_33_serial;	
	delete [] array_34_serial; 	delete [] array_35_serial;	delete [] array_36_serial;	
	delete [] array_37_serial;	delete [] array_38_serial; delete [] array_39_serial;
	delete [] array_40_serial;
	delete [] array_41_serial;	delete [] array_42_serial;	delete [] array_43_serial;	
	delete [] array_44_serial; 	delete [] array_45_serial;	delete [] array_46_serial;
	delete [] array_47_serial;	delete [] array_48_serial; delete [] array_49_serial;
	delete [] array_50_serial;
	delete [] array_51_serial;	delete [] array_52_serial;	delete [] array_53_serial;	
	delete [] array_54_serial; 	delete [] array_55_serial;	delete [] array_56_serial;
	delete [] array_57_serial;	delete [] array_58_serial; delete [] array_59_serial;
	delete [] array_60_serial;
	delete [] array_61_serial;	delete [] array_62_serial;	delete [] array_63_serial;	
	delete [] array_64_serial; 	delete [] array_65_serial;	delete [] array_66_serial;
	delete [] array_67_serial;	delete [] array_68_serial; delete [] array_69_serial;
	delete [] array_70_serial;
	
	delete [] c_serial;
	
	printf("End of program!\n");
	
	
}